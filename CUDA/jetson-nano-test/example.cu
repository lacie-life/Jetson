#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void AplusB(int *ret, int a, int b){
    //printf("hello\n");
    ret[threadIdx.x] = a + b + threadIdx.x;
}

int main() {
    int *ret;

    hipMalloc( (void**)&ret, 1000*sizeof(int));

    AplusB<<<1, 1000>>>(ret, 10, 100);

    if (hipPeekAtLastError() != hipSuccess) {
        printf("add start exceuting! \n");
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
        printf("\n");
    }

    int *host_ret = (int *)malloc(1000*sizeof(int));

    hipMemcpy(host_ret, ret, 1000*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < 10; i++){
        printf("%d: A+B = %d \n", i, host_ret[i]);
    }

    free(host_ret);
    hipFree(ret);
    return 0;
}

