#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Multi Device Cooperative Groups, also uses Unified Memory optimized using
 * prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <map>
#include <iostream>
#include <set>
#include <utility>

#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

const char *sSDKname = "conjugateGradientMultiDeviceCG";

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

__device__ double grid_dot_result = 0.0;

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz) {
  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = static_cast<float>(rand()) / RAND_MAX + 10.0f;
  val[1] = static_cast<float>(rand()) / RAND_MAX;
  int start;

  for (int i = 1; i < N; i++) {
    if (i > 1) {
      I[i] = I[i - 1] + 3;
    } else {
      I[1] = 2;
    }

    start = (i - 1) * 3 + 2;
    J[start] = i - 1;
    J[start + 1] = i;

    if (i < N - 1) {
      J[start + 2] = i + 1;
    }

    val[start] = val[start - 1];
    val[start + 1] = static_cast<float>(rand()) / RAND_MAX + 10.0f;

    if (i < N - 1) {
      val[start + 2] = static_cast<float>(rand()) / RAND_MAX;
    }
  }

  I[N] = nz;
}

// I - contains location of the given non-zero element in the row of the matrix
// J - contains location of the given non-zero element in the column of the
// matrix val - contains values of the given non-zero elements of the matrix
// inputVecX - input vector to be multiplied
// outputVecY - resultant vector
void cpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
             float *inputVecX, float *outputVecY) {
  for (int i = 0; i < num_rows; i++) {
    int num_elems_this_row = I[i + 1] - I[i];

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[I[i] + j] * inputVecX[J[I[i] + j]];
    }
    outputVecY[i] = output;
  }

  return;
}

double dotProduct(float *vecA, float *vecB, int size) {
  double result = 0.0;

  for (int i = 0; i < size; i++) {
    result = result + (vecA[i] * vecB[i]);
  }

  return result;
}

void scaleVector(float *vec, float alpha, int size) {
  for (int i = 0; i < size; i++) {
    vec[i] = alpha * vec[i];
  }
}

void saxpy(float *x, float *y, float a, int size) {
  for (int i = 0; i < size; i++) {
    y[i] = a * x[i] + y[i];
  }
}

void cpuConjugateGrad(int *I, int *J, float *val, float *x, float *Ax, float *p,
                      float *r, int nnz, int N, float tol) {
  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, b, a, na;

  cpuSpMV(I, J, val, nnz, N, alpha, x, Ax);
  saxpy(Ax, r, alpham1, N);

  float r1 = dotProduct(r, r, N);

  int k = 1;

  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      scaleVector(p, b, N);

      saxpy(r, p, alpha, N);
    } else {
      for (int i = 0; i < N; i++) p[i] = r[i];
    }

    cpuSpMV(I, J, val, nnz, N, alpha, p, Ax);

    float dot = dotProduct(p, Ax, N);
    a = r1 / dot;

    saxpy(p, x, a, N);
    na = -a;
    saxpy(Ax, r, na, N);

    r0 = r1;
    r1 = dotProduct(r, r, N);

    printf("\nCPU code iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }
}

__device__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows,
                        float alpha, float *inputVecX, float *outputVecY,
                        cg::thread_block &cta,
                        const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < num_rows; i += multi_grid.size()) {
    int row_elem = I[i];
    int next_row_elem = I[i + 1];
    int num_elems_this_row = next_row_elem - row_elem;

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
    }

    outputVecY[i] = output;
  }
}

__device__ void gpuSaxpy(float *x, float *y, float a, int size,
                         const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    y[i] = a * x[i] + y[i];
  }
}

__device__ void gpuDotProduct(float *vecA, float *vecB, int size,
                              const cg::thread_block &cta,
                              const cg::multi_grid_group &multi_grid) {
  extern __shared__ double tmp[];

  double temp_sum = 0.0;

  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    temp_sum += static_cast<double>(vecA[i] * vecB[i]);
  }

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

  if (tile32.thread_rank() == 0) {
    tmp[tile32.meta_group_rank()] = temp_sum;
  }

  cg::sync(cta);

  if (tile32.meta_group_rank() == 0) {
     temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
     temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
      atomicAdd(&grid_dot_result, temp_sum);
    }
  }
}

__device__ void gpuCopyVector(float *srcA, float *destB, int size,
                              const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    destB[i] = srcA[i];
  }
}

__device__ void gpuScaleVectorAndSaxpy(float *x, float *y, float a, float scale, int size,
                         const cg::multi_grid_group &multi_grid) {
  for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
    y[i] = a * x[i] + scale * y[i];
  }
}

extern "C" __global__ void multiGpuConjugateGradient(
    int *I, int *J, float *val, float *x, float *Ax, float *p, float *r,
    double *dot_result, int nnz, int N, float tol) {
  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();
  cg::multi_grid_group multi_grid = cg::this_multi_grid();

  const int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, r1, b, a, na;

  for (int i = multi_grid.thread_rank(); i < N; i += multi_grid.size()) {
    r[i] = 1.0;
    x[i] = 0.0;
  }

  cg::sync(grid);

  gpuSpMV(I, J, val, nnz, N, alpha, x, Ax, cta, multi_grid);

  cg::sync(grid);

  gpuSaxpy(Ax, r, alpham1, N, multi_grid);

  cg::sync(grid);

  gpuDotProduct(r, r, N, cta, multi_grid);

  cg::sync(grid);

  if (grid.thread_rank() == 0) {
    atomicAdd_system(dot_result, grid_dot_result);
    grid_dot_result = 0.0;
  }
  cg::sync(multi_grid);

  r1 = *dot_result;

  int k = 1;
  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      gpuScaleVectorAndSaxpy(r, p, alpha, b, N, multi_grid);
    } else {
      gpuCopyVector(r, p, N, multi_grid);
    }

    cg::sync(multi_grid);

    gpuSpMV(I, J, val, nnz, N, alpha, p, Ax, cta, multi_grid);

    if (multi_grid.thread_rank() == 0) {
      *dot_result = 0.0;
    }
    cg::sync(multi_grid);

    gpuDotProduct(p, Ax, N, cta, multi_grid);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
      atomicAdd_system(dot_result, grid_dot_result);
      grid_dot_result = 0.0;
    }
    cg::sync(multi_grid);

    a = r1 / *dot_result;

    gpuSaxpy(p, x, a, N, multi_grid);

    na = -a;

    gpuSaxpy(Ax, r, na, N, multi_grid);

    r0 = r1;

    cg::sync(multi_grid);
    if (multi_grid.thread_rank() == 0) {
      *dot_result = 0.0;
    }

    cg::sync(multi_grid);

    gpuDotProduct(r, r, N, cta, multi_grid);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
      atomicAdd_system(dot_result, grid_dot_result);
      grid_dot_result = 0.0;
    }
    cg::sync(multi_grid);

    r1 = *dot_result;
    k++;
  }
}

// Map of device version to device number
std::multimap<std::pair<int, int>, int> getIdenticalGPUs() {
  int numGpus = 0;
  checkCudaErrors(hipGetDeviceCount(&numGpus));

  std::multimap<std::pair<int, int>, int> identicalGpus;

  for (int i = 0; i < numGpus; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));

    // Filter unsupported devices
    if (deviceProp.cooperativeMultiDeviceLaunch &&
        deviceProp.concurrentManagedAccess) {
      identicalGpus.emplace(std::make_pair(deviceProp.major, deviceProp.minor), i);
    }
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i,
          deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  return identicalGpus;
}

int main(int argc, char **argv) {
  constexpr size_t kNumGpusRequired = 2;
  int N = 0, nz = 0, *I = NULL, *J = NULL;
  float *val = NULL;
  const float tol = 1e-5f;
  float *x;
  float rhs = 1.0;
  float r1;
  float *r, *p, *Ax;

  printf("Starting [%s]...\n", sSDKname);
  auto gpusByArch = getIdenticalGPUs();

  auto it = gpusByArch.begin();
  auto end = gpusByArch.end();

  auto bestFit = std::make_pair(it, it);
  // use std::distance to find the largest number of GPUs amongst architectures
  auto distance = [](decltype(bestFit) p){return std::distance(p.first, p.second);};

  // Read each unique key/pair element in order
  for (; it != end; it = gpusByArch.upper_bound(it->first)) {
    // first and second are iterators bounded within the architecture group
    auto testFit = gpusByArch.equal_range(it->first);
    // Always use devices with highest architecture version or whichever has the most devices available
    if (distance(bestFit) <= distance(testFit))
        bestFit = testFit;
  }

  if (distance(bestFit) < kNumGpusRequired) {
    printf(
        "No Two or more GPUs with same architecture capable of "
        "cooperativeMultiDeviceLaunch & concurrentManagedAccess found. "
        "\nWaiving the sample\n");
    exit(EXIT_WAIVED);
  }

  std::set<int> bestFitDeviceIds;

  // check & select peer-to-peer access capable GPU devices as enabling p2p access between participating
  // GPUs gives better performance for multi_grid sync.
  for (auto itr = bestFit.first; itr != bestFit.second; itr++) {
    int deviceId = itr->second;
    checkCudaErrors(hipSetDevice(deviceId));

    std::for_each(itr, bestFit.second, [&deviceId, &bestFitDeviceIds](decltype(*itr) mapPair) {
      if (deviceId != mapPair.second)
      {
        int access = 0;
        checkCudaErrors(hipDeviceCanAccessPeer(&access, deviceId, mapPair.second));
        printf("Device=%d %s Access Peer Device=%d\n", deviceId, access ? "CAN" : "CANNOT", mapPair.second);
        if (access && bestFitDeviceIds.size() < kNumGpusRequired) {
          bestFitDeviceIds.emplace(deviceId);
          bestFitDeviceIds.emplace(mapPair.second);
        }
        else {
          printf("Ignoring device %i (max devices exceeded)\n", mapPair.second);
        }
      }
    });

    if (bestFitDeviceIds.size() >= kNumGpusRequired)
    {
      printf("Selected p2p capable devices - ");
      for (auto devicesItr = bestFitDeviceIds.begin(); devicesItr != bestFitDeviceIds.end(); devicesItr++)
      {
        printf("deviceId = %d  ", *devicesItr);
      }
      printf("\n");
      break;
    }
  }

  // if bestFitDeviceIds.size() == 0 it means the GPUs in system are not p2p capable,
  // hence we add it without p2p capability check.
  if (!bestFitDeviceIds.size())
  {
    printf("Devices involved are not p2p capable.. selecting %zu of them\n", kNumGpusRequired);
    std::for_each(bestFit.first, bestFit.second, [&bestFitDeviceIds](decltype(*bestFit.first) mapPair) {
      if (bestFitDeviceIds.size() < kNumGpusRequired) {
        bestFitDeviceIds.emplace(mapPair.second);
      }
      else {
        printf("Ignoring device %i (max devices exceeded)\n", mapPair.second);
      }
      // Insert the sequence into the deviceIds set
    });
  }
  else
  {
    // perform hipDeviceEnablePeerAccess in both directions for all participating devices
    // of a hipLaunchCooperativeKernelMultiDevice call this gives better performance for multi_grid sync.
    for (auto p1_itr = bestFitDeviceIds.begin(); p1_itr != bestFitDeviceIds.end(); p1_itr++)
    {
      checkCudaErrors(hipSetDevice(*p1_itr));
      for (auto p2_itr = bestFitDeviceIds.begin(); p2_itr != bestFitDeviceIds.end(); p2_itr++)
      {
        if (*p1_itr != *p2_itr)
        {
          checkCudaErrors(hipDeviceEnablePeerAccess(*p2_itr, 0 ));
          checkCudaErrors(hipSetDevice(*p1_itr));
        }
      }
    }
  }

  /* Generate a random tridiagonal symmetric matrix in CSR format */
  N = 10485760 * 2;
  nz = (N - 2) * 3 + 4;

  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&I), sizeof(int) * (N + 1)));
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&J), sizeof(int) * nz));
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&val), sizeof(float) * nz));

  float *val_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * nz));

  genTridiag(I, J, val_cpu, N, nz);

  memcpy(val, val_cpu, sizeof(float) * nz);
  checkCudaErrors(
      hipMemAdvise(I, sizeof(int) * (N + 1), hipMemAdviseSetReadMostly, 0));
  checkCudaErrors(
      hipMemAdvise(J, sizeof(int) * nz, hipMemAdviseSetReadMostly, 0));
  checkCudaErrors(
      hipMemAdvise(val, sizeof(float) * nz, hipMemAdviseSetReadMostly, 0));

  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&x), sizeof(float) * N));

  double *dot_result;
  checkCudaErrors(hipMallocManaged(reinterpret_cast<void **>(&dot_result),
                                    sizeof(double)));

  checkCudaErrors(hipMemset(dot_result, 0.0, sizeof(double)));

  // temp memory for ConjugateGradient
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&r), N * sizeof(float)));
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&p), N * sizeof(float)));
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&Ax), N * sizeof(float)));

  std::cout << "\nRunning on GPUs = " << kNumGpusRequired << std::endl;
  hipStream_t nStreams[kNumGpusRequired];

  int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK/32) + 1);
  int numBlocksPerSm = INT_MAX;
  int numThreads = THREADS_PER_BLOCK;
  int numSms = INT_MAX;
  auto deviceId = bestFitDeviceIds.begin();

  // set numSms & numBlocksPerSm to be lowest of 2 devices
  while (deviceId != bestFitDeviceIds.end()) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, *deviceId));

    int numBlocksPerSm_current=0;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, multiGpuConjugateGradient, numThreads, sMemSize));

    if (numBlocksPerSm > numBlocksPerSm_current)
    {
        numBlocksPerSm = numBlocksPerSm_current;
    }
    if (numSms > deviceProp.multiProcessorCount)
    {
        numSms = deviceProp.multiProcessorCount;
    }
    deviceId++;
  }

  if (!numBlocksPerSm) {
    printf(
        "Max active blocks per SM is returned as 0.\n Hence, Waiving the "
        "sample\n");
    exit(EXIT_WAIVED);
  }

  int device_count = 0;
  int totalThreadsPerGPU = numSms * numBlocksPerSm * THREADS_PER_BLOCK;
  deviceId =  bestFitDeviceIds.begin();;
  while (deviceId != bestFitDeviceIds.end()) {
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipStreamCreate(&nStreams[device_count]));

    int perGPUIter = N / (totalThreadsPerGPU * kNumGpusRequired);
    int offset_Ax = device_count * totalThreadsPerGPU;
    int offset_r = device_count * totalThreadsPerGPU;
    int offset_p = device_count * totalThreadsPerGPU;
    int offset_x = device_count * totalThreadsPerGPU;

    checkCudaErrors(hipMemPrefetchAsync(I, sizeof(int) * N, *deviceId,
                                         nStreams[device_count]));
    checkCudaErrors(hipMemPrefetchAsync(val, sizeof(float) * nz, *deviceId,
                                         nStreams[device_count]));
    checkCudaErrors(hipMemPrefetchAsync(J, sizeof(float) * nz, *deviceId,
                                         nStreams[device_count]));

    if (offset_Ax <= N) {
      for (int i = 0; i < perGPUIter; i++) {
        hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);
        hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);
        hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);
        hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);

        hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);
        hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);
        hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);
        hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);

        offset_Ax += totalThreadsPerGPU * kNumGpusRequired;
        offset_r += totalThreadsPerGPU * kNumGpusRequired;
        offset_p += totalThreadsPerGPU * kNumGpusRequired;
        offset_x += totalThreadsPerGPU * kNumGpusRequired;

        if (offset_Ax >= N) {
          break;
        }
      }
    }

    device_count++;
    deviceId++;
  }

#if ENABLE_CPU_DEBUG_CODE
  float *Ax_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  float *r_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  float *p_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  float *x_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));

  for (int i = 0; i < N; i++) {
    r_cpu[i] = 1.0;
    Ax_cpu[i] = x_cpu[i] = 0.0;
  }
#endif

  printf("Total threads per GPU = %d numBlocksPerSm  = %d\n",
         numSms * numBlocksPerSm * THREADS_PER_BLOCK, numBlocksPerSm);
  dim3 dimGrid(numSms * numBlocksPerSm, 1, 1), dimBlock(THREADS_PER_BLOCK, 1, 1);
  void *kernelArgs[] = {
      (void *)&I,  (void *)&J, (void *)&val, (void *)&x,
      (void *)&Ax, (void *)&p, (void *)&r,   (void *)&dot_result,
      (void *)&nz, (void *)&N, (void *)&tol,
  };
  hipLaunchParams *launchParamsList = (hipLaunchParams *)malloc(
      sizeof(hipLaunchParams) * kNumGpusRequired);
  for (int i = 0; i < kNumGpusRequired; i++) {
    launchParamsList[i].func = (void *)multiGpuConjugateGradient;
    launchParamsList[i].gridDim = dimGrid;
    launchParamsList[i].blockDim = dimBlock;
    launchParamsList[i].sharedMem = sMemSize;
    launchParamsList[i].stream = nStreams[i];
    launchParamsList[i].args = kernelArgs;
  }

  printf("Launching kernel\n");

  checkCudaErrors(hipLaunchCooperativeKernelMultiDevice(
      launchParamsList, kNumGpusRequired,
      hipCooperativeLaunchMultiDeviceNoPreSync |
          hipCooperativeLaunchMultiDeviceNoPostSync));

  checkCudaErrors(
      hipMemPrefetchAsync(x, sizeof(float) * N, hipCpuDeviceId));
  checkCudaErrors(
        hipMemPrefetchAsync(dot_result, sizeof(double), hipCpuDeviceId));

  deviceId =  bestFitDeviceIds.begin();;
  device_count = 0;
  while (deviceId != bestFitDeviceIds.end()) {
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipStreamSynchronize(nStreams[device_count++]));
    deviceId++;
  }

  r1 = *dot_result;

  printf("GPU Final, residual = %e \n  ", sqrt(r1));

#if ENABLE_CPU_DEBUG_CODE
  cpuConjugateGrad(I, J, val, x_cpu, Ax_cpu, p_cpu, r_cpu, nz, N, tol);
#endif

  float rsum, diff, err = 0.0;

  for (int i = 0; i < N; i++) {
    rsum = 0.0;

    for (int j = I[i]; j < I[i + 1]; j++) {
      rsum += val_cpu[j] * x[J[j]];
    }

    diff = fabs(rsum - rhs);

    if (diff > err) {
      err = diff;
    }
  }

  checkCudaErrors(hipFree(I));
  checkCudaErrors(hipFree(J));
  checkCudaErrors(hipFree(val));
  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(r));
  checkCudaErrors(hipFree(p));
  checkCudaErrors(hipFree(Ax));
  checkCudaErrors(hipFree(dot_result));
  free(val_cpu);

#if ENABLE_CPU_DEBUG_CODE
  free(Ax_cpu);
  free(r_cpu);
  free(p_cpu);
  free(x_cpu);
#endif

  printf("Test Summary:  Error amount = %f \n", err);
  fprintf(stdout, "&&&& conjugateGradientMultiDeviceCG %s\n",
          (sqrt(r1) < tol) ? "PASSED" : "FAILED");
  exit((sqrt(r1) < tol) ? EXIT_SUCCESS : EXIT_FAILURE);
}
