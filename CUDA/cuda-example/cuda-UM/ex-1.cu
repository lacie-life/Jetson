
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printme(char *str){
    printf(str);
}

int main(){
    // Allocate 100 bytes of memory, accessible to both Host and Device code
    char *s;
    hipMallocManaged(&s, 100);
    // Note direct Host-code use of "s"
    strncpy(s, "Hello Unified Memory\n", 99);
    // Here we pass "s" to a kernel without explicitly copying
    printme<<< 1, 1 >>>(s);
    hipDeviceSynchronize();
    // Free as for normal CUDA allocations
    hipFree(s);
    return 0;
}