#include "hip/hip_runtime.h"
#include <stdio.h>

#define TPB 64

void dotLauncher(int *res, const int *a, const int *b, int n){
    int *d_res;
    int *d_a;
    int *d_b;

    hipMalloc(&d_res, sizeof(int));
    hipMalloc(&d_a, n*sizeof(int));
    hipMalloc(&d_b, n*sizeof(int));

    hipMemset(d_res, 0, sizeof(int));
    hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);

    dotKernel<<<(n+TPB-1)/TPB, TPB>>>(d_res, d_a, d_b, n);

    hipMemcpy(res, d_res, sizeof(int), cudaMemcpyDiviceToHost);

    hipFree(d_res);
    hipFree(d_b);
    hipFree(d_a);
}


__global__ void dotKernel(int *d_res, const int *d_a, const int *d_b, int n){

    const int idx = threadIdx.x + blockDim.x*blockIdx;
    if(idx>=n) return;

    const int s_idx = threadIdx.x;

    __shared__ int s_prod[TPB];
    s_prod[s_idx] = d_a[idx]*d_b[idx];
    __syncthread();

    if(s_idx == 0){
        int blocSum = 0;
        for(int j = 0; j < blockDim.x; ++j){
            blocSum += s_prod[j];
        }
        printf("Block_%d, blockSum = %d \n", blockIdx.x, blockSum);
        aomicAdd(d_res, blockSum);
    }
}

int main(){
    
}