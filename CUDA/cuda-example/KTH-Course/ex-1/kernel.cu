
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 64
#define TPB 32

__device__ float scale(int i, int n){
    return ((float)i)/(n -1);
}

__device__ float distance(float x1, float x2){
    return sqrt((x2 - x1)*(x2 - x1));
}

__global__ void distanceKernel(float *d_out, float ref, float len){
    //
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = scale(i, len);
    d_out[i] = distance(x, ref);
    printf("i = %2d: dist from %f to %f is %f \n", i, ref, x, d_out[i]);
}

int main(){

    const float ref = 0.5;

    float *d_out = 0;
    hipMalloc(&d_out, N*sizeof(float));

    distanceKernel<<<N/TPB, TPB>>>(d_out, ref, N);

    // cudaDeviceSynchronize()
    hipFree(d_out);
    return 0;
}