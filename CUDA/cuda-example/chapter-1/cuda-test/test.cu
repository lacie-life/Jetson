#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#define N  5

__global__ void add(int *a, int *b, int *c)
{
        printf("after copy ....\n");
        for (int i = 0; i<N; i++)
        {
                printf("%d + %d ? \n", a[i], b[i]);
        }
        int tid = blockIdx.x;
        if (tid<N){
            c[tid] = a[tid] + b[tid];
            printf("Result: %d + %d = %d \n",a[tid], b[tid], c[tid]);
        }  
}

int main(void)
{
        int a[N],b[N],c[N];
        int *dev_a, *dev_b, *dev_c;

        hipError_t rc = hipMalloc((void **) &dev_a, N*sizeof(int));

        if (rc != hipSuccess)
            printf("Could not allocate memory: %d", rc);

        hipMalloc((void**)&dev_b, N * sizeof(int));
        hipMalloc((void**)&dev_c, N * sizeof(int));

        for (int i = 0; i<N; i++)
        {
                a[i] = i;
                b[i] = i;
                printf("%d + %d ? \n", a[i], b[i]);
        }

        hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
        
        add<<<N,1>>>(dev_a, dev_b, dev_c);
        hipDeviceSynchronize();
        
        if (hipPeekAtLastError() != hipSuccess) {
            printf("add start exceuting! \n");
            printf("The error is %s", hipGetErrorString(hipGetLastError()));
            printf("\n");
        }
        
        hipError_t err2 = hipMemcpy( c, dev_c, sizeof(int), hipMemcpyDeviceToHost );
        if(err2 != hipSuccess){
            printf("The error is %s", hipGetErrorString(err2));
            printf("\n");
        }

        for(int i =0; i<N; i++)
            printf("%d + %d = %d\n",a[i],b[i],c[i]);

        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);

        return EXIT_SUCCESS;
}