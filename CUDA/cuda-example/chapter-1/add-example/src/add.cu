#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "../../../common/book.h"
#include <stdio.h>

__global__ void add( int a, int b, int *c ) {
    printf("Hello from GPU !!! \n");
    *c = a + b;
}

int main( void ) {
    int c;
    int *dev_c;

    hipMalloc( (void**)&dev_c, sizeof(int));

    add<<<1,1>>>( 2, 7, dev_c );

    if (hipPeekAtLastError() != hipSuccess) {
        printf("add start exceuting! \n");
        printf("The error is %s", hipGetErrorString(hipGetLastError()));
        printf("\n");
    }

    hipMemcpy( &c,
                dev_c,
                sizeof(int),
                hipMemcpyDeviceToHost );
    
    printf( "2 + 7 = %d\n", c );

    hipFree( dev_c );

    return 0;
}