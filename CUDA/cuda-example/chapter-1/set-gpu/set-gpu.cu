#include "../../common/book.h"

int main( void ) {
    hipDeviceProp_t  prop;
    int dev;

    HANDLE_ERROR( hipGetDevice( &dev ) );
    printf( "ID of current CUDA device:  %d\n", dev );

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

    HANDLE_ERROR( hipSetDevice( dev ) );
}